
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cstdio>            // printf
#include <cstdlib>           // EXIT_FAILURE
#include <iostream>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        std::printf("CUDA API failed at line %d with error: %s (%d)\n",        \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}                                                                              \


#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, \
             hipsparseGetErrorString(status), status);                          \
    }                                                                          \
  }


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");

}


int main() {
  //  cuda_hello<<<1,1>>>(); 

  
  
  int   A_num_rows   = 3; // [5,4]  A^T = [4, 5]
  int   A_num_cols   = 2;
  bool  transpose_a  = true;
  int   A_nnz        = 4;
  int   B_num_rows   = 3; // [5, 3]
  int   B_num_cols   = 3;

/*   if(transpose_a) {
    std::swap(A_num_rows, A_num_cols);
  } */ 
  int   ldb          = B_num_rows; // 5
  int   ldc          = transpose_a? A_num_rows: A_num_cols; // 4
  int   B_size       = 3 * 3;  // 5 * 3 
  int   C_size       = 2 * 3; //  4 * 3  [4, 5][5, 3]
  //int   hA_rows[]    = { 0, 0, 0, 1, 2, 2, 2, 3, 3 };
  int   hA_rows[]    = { 0, 1, 3, 4 };
  
  int   hA_columns[] = { 0, 0, 1, 1 };
  float hA_values[]  = { 1, 2, 3, 4 };
  float  hB[]        = { 1,  1,  1, 1, 0, 1, 0, 0, 1 };
  float  hC[]        = { 0.0f, 0.0f, 0.0f, 0.0f,
                         0.0f, 0.0f, 0.0f, 0.0f,
                         0.0f, 0.0f};
  float  hC_result[] = { 19.0f,  8.0f,  51.0f,  52.0f,
                         43.0f, 24.0f, 123.0f, 120.0f,
                         67.0f, 40.0f, 195.0f, 188.0f };
  float  alpha       = 1.0f;
  float  beta        = 0.0f;
  //--------------------------------------------------------------------------
  // Device memory management
  int   *dA_rows, *dA_columns;
  float *dA_values, *dB, *dC;
  CHECK_CUDA( hipMalloc((void**) &dA_rows,    (A_num_rows+1)*sizeof(int)))
  CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))    )
  CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))  )
  CHECK_CUDA( hipMalloc((void**) &dB,         B_size * sizeof(float)) )
  CHECK_CUDA( hipMalloc((void**) &dC,         C_size * sizeof(float)) )

  CHECK_CUDA( hipMemcpy(dA_rows, hA_rows, (A_num_rows+1) * sizeof(int),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float),
                         hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(float),
                         hipMemcpyHostToDevice) )
  //--------------------------------------------------------------------------
  //std::swap(A_num_rows, A_num_cols);
   int b_row = transpose_a? A_num_rows: A_num_cols;
   int c_row = transpose_a? A_num_cols: A_num_rows;
   int bc_col = B_num_cols;
  // CUSPARSE APIs
  hipsparseHandle_t     handle = NULL;
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  void*                dBuffer    = NULL;
  size_t               bufferSize = 0;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )
  // Create sparse matrix A in COO format
  auto a_row = A_num_rows, a_col = A_num_cols;
  if(transpose_a) std::swap(a_row, a_col);
  std::cout << a_row << ' ' << a_col << std::endl;
  CHECK_CUSPARSE( hipsparseCreateCsr(&matA, 3, 2, A_nnz,
                                    dA_rows, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
  // Create dense matrix B
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, 3, 3, /*ld=*/3, dB,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  // Create dense matrix C
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, 2, 3, /*ld=*/3, dC,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  // allocate an external buffer if needed
  CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                               handle,
                               HIPSPARSE_OPERATION_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC, HIP_R_32F,
                               HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
  CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
  std::printf("buffer size %d\n", bufferSize);

/*   A [3,2] sparse,  A^T [2,3] csr
  X [3,6] dense
  Y =  op(A) op(X)    spmm
  op = transpose(), non_transpose
  Y = op(A) X = A^T X = [2, 3] [3, 6] = [2, 6]
  row: a11 a12 
  col: a11 a21 */
  // execute SpMM
  CHECK_CUSPARSE( hipsparseSpMM(handle,
                               HIPSPARSE_OPERATION_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC, HIP_R_32F,
                               HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
  CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
  CHECK_CUSPARSE( hipsparseDestroy(handle) )
  //--------------------------------------------------------------------------
  // device result check
  CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(float),
                         hipMemcpyDeviceToHost) )

  for(int i = 0; i < 2; i++) { //row
      for(int j = 0; j < 3; j++) { //col
        std::printf(" %f ", hC[i*2 + j]);
      }
      std::printf("\n");
    }


    hipFree(dA_rows);
    hipFree(dA_columns);
    hipFree(dA_values);
    hipFree(dB);
    hipFree(dC);
    printf("Hello World from CPU!\n");
    hipDeviceSynchronize();
    return 0;
}