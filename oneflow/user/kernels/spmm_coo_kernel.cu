#include "oneflow/core/framework/framework.h"
#include "oneflow/core/device/cuda_util.h"

#include <hipsparse.h>

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, \
             hipsparseGetErrorString(status), status);                          \
    }                                                                          \
  }

namespace oneflow {


template<typename T>
class CudaSpmmCooKernel final : public user_op::OpKernel {
  public:
  CudaSpmmCooKernel() = default;
  ~CudaSpmmCooKernel() = default;

  private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const int64_t A_num_rows = ctx->Attr<int64_t>("a_rows");
    const int64_t A_num_cols = ctx->Attr<int64_t>("a_cols");

    const user_op::Tensor *a_cooRowInd = ctx->Tensor4ArgNameAndIndex("a_cooRowInd", 0);
    const user_op::Tensor *a_cooColInd = ctx->Tensor4ArgNameAndIndex("a_cooColInd", 0);
    const user_op::Tensor *a_cooValues = ctx->Tensor4ArgNameAndIndex("a_cooValues", 0);
    const user_op::Tensor *b = ctx->Tensor4ArgNameAndIndex("b", 0);

    user_op::Tensor *out_tensor = ctx->Tensor4ArgNameAndIndex("out", 0);

    const int32_t *a_cooRowInd_ptr = a_cooRowInd->dptr<int32_t>();
    const int32_t *a_cooColInd_ptr = a_cooColInd->dptr<int32_t>();
    const float *a_cooValues_ptr = a_cooValues->dptr<float>();
    const float *b_ptr = b->dptr<float>();


    int A_nnz = a_cooRowInd->shape_view().elem_cnt();
    int B_num_rows = b->shape_view().At(0);
    int B_num_cols = b->shape_view().At(1); 

    std::cout << "in cuda kernel: " << A_nnz << ' ' << B_num_rows << ' ' << B_num_cols << '\n';
    int ldb = B_num_cols;
    int ldc = B_num_cols;
    int B_size = B_num_rows * B_num_cols;
    int C_size = A_num_rows * B_num_cols;

    const int32_t *hA_rows = a_cooRowInd_ptr;
    const int32_t *hA_columns = a_cooColInd_ptr;
    const float *hA_values = a_cooValues_ptr;
    const float *hB = b_ptr;
    float *hC = out_tensor->mut_dptr<float>();

    float alpha = 1.0f;
    float beta = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int32_t *dA_rows, *dA_columns;
    float *dA_values, *dB, *dC;
    OF_CUDA_CHECK(hipMalloc((void **)&dA_rows, A_nnz * sizeof(int32_t)));
    OF_CUDA_CHECK(hipMalloc((void **)&dA_columns, A_nnz * sizeof(int32_t)));
    OF_CUDA_CHECK(hipMalloc((void **)&dA_values, A_nnz * sizeof(float)));
    OF_CUDA_CHECK(hipMalloc((void **)&dB, B_size * sizeof(float)));
    OF_CUDA_CHECK(hipMalloc((void **)&dC, C_size * sizeof(float)));

    OF_CUDA_CHECK(hipMemcpy(dA_rows, hA_rows, A_nnz * sizeof(int32_t), 
                             hipMemcpyHostToDevice));
    OF_CUDA_CHECK(
        hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int32_t),
                   hipMemcpyHostToDevice));
    OF_CUDA_CHECK(hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                             hipMemcpyHostToDevice));
    OF_CUDA_CHECK(hipMemcpy(dB, hB, B_size * sizeof(float),
                             hipMemcpyHostToDevice));
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle))
    // Create sparse matrix A in COO format
    CHECK_CUSPARSE(hipsparseCreateCoo(&matA, A_num_rows, A_num_cols, A_nnz, dA_rows, dA_columns,
                                      dA_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                                      HIP_R_32F))
    // Create dense matrix B
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, B_num_rows, B_num_cols, ldb, dB,
                                       HIP_R_32F, HIPSPARSE_ORDER_ROW))
    // Create dense matrix C
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, hC,
                                       HIP_R_32F,HIPSPARSE_ORDER_ROW))
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA,
                                matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize))
    OF_CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    // execute SpMM
    CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC,
                                HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer))

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))
    CHECK_CUSPARSE(hipsparseDestroy(handle))
    //--------------------------------------------------------------------------
    // device memory deallocation
    OF_CUDA_CHECK(hipFree(dBuffer));
    OF_CUDA_CHECK(hipFree(dA_rows));
    OF_CUDA_CHECK(hipFree(dA_columns));
    OF_CUDA_CHECK(hipFree(dA_values));
    OF_CUDA_CHECK(hipFree(dB));
    OF_CUDA_CHECK(hipFree(dC));
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_CUDA_SPMM_COO_KERNEL(dtype)                                                   \
  REGISTER_USER_KERNEL("spmm_coo")                                                             \
      .SetCreateFn<CudaSpmmCooKernel<dtype>>()                                                \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                       \
                        && (user_op::HobDataType("out", 0) == GetDataType<dtype>::value));
/*   \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) -> size_t {                          \
        const Shape& in_shape = ctx->InputShape("b", 0);                                 \
        const int32_t instance_size = in_shape.elem_cnt();                                   \
        size_t sort_tmp_buffer_bytes =                                                       \
            InferTempStorageForSortKeysAscending<dtype>(1, instance_size);                   \
        size_t sort_tensor_buffer_bytes = GetCudaAlignedSize(instance_size * sizeof(dtype)); \
        return sort_tmp_buffer_bytes + sort_tensor_buffer_bytes;                             \
      })
      */
REGISTER_CUDA_SPMM_COO_KERNEL(float);
REGISTER_CUDA_SPMM_COO_KERNEL(double);
REGISTER_CUDA_SPMM_COO_KERNEL(int8_t);
REGISTER_CUDA_SPMM_COO_KERNEL(uint8_t);
REGISTER_CUDA_SPMM_COO_KERNEL(int32_t);
REGISTER_CUDA_SPMM_COO_KERNEL(int64_t);

}  // namespace oneflow