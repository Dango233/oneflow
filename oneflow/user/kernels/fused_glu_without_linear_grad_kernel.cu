#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/common/device_type.h"
#include "oneflow/core/ep/include/primitive/matmul.h"
#include "oneflow/core/ep/include/primitive/unary_op.h"
#include "oneflow/core/ep/common/primitive/unary_functor.h"
#include "oneflow/core/ep/cuda/primitive/unary_functor.cuh"

#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif  // CUDA_VERSION >= 11000
#include "oneflow/core/device/cuda_pseudo_bfloat16.h"

namespace oneflow {

namespace {

// current: pack_size default to be 1, pack_num default to be number of elements
template<typename T, typename IndexType, typename FUNCTOR, ep::primitive::UnaryOp act_type, int32_t pack_size>
__global__ void FusedGluWithoutLinearGradGpu(
    const IndexType m, const IndexType packed_n, const IndexType pack_num, 
    const IndexType input_stride, const IndexType output_stride,
    FUNCTOR act_grad_functor,
    ep::primitive::UnaryFunctor<DeviceType::kCUDA, act_type, T, T> act,
    const T* dy, const T* matmul_wx, const T* matmul_vx, 
    T* d_matmul_wx, T* d_matmul_vx
){
    // obtain global thread index
    IndexType global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // workload of current thread
    for(IndexType pack_index = global_thread_id, step = gridDim.x * blockDim.x;
      pack_index < pack_num; pack_index += step){
        // TODO
    }
}

template<typename T, ep::primitive::UnaryOp act_type>
void DispatchAlignment(ep::Stream* stream, const int64_t m, const int64_t n, 
  const int64_t input_stride, const int64_t output_stride,
  const T* dy, const T* matmul_wx, const T* matmul_vx, 
    T* d_matmul_wx, T* d_matmul_vx
) {
    const auto IsAligned = [&](const size_t alignment) {
    const uintptr_t dy_ptr = reinterpret_cast<uintptr_t>(dy);
    const uintptr_t matmul_wx_ptr = reinterpret_cast<uintptr_t>(matmul_wx);
    const uintptr_t matmul_vx_ptr = reinterpret_cast<uintptr_t>(matmul_vx);
    const uintptr_t d_matmul_wx_ptr = reinterpret_cast<uintptr_t>(d_matmul_wx);
    const uintptr_t d_matmul_vx_ptr = reinterpret_cast<uintptr_t>(d_matmul_vx);

    return (/* memory address alignment */
            dy_ptr % alignment == 0 && matmul_vx_ptr % alignment == 0
            && matmul_wx_ptr % alignment == 0 && d_matmul_wx_ptr % alignment == 0
            && d_matmul_vx_ptr % alignment == 0
            /* #element per row alignment */
            && n % (alignment / sizeof(T)) == 0);
    };
}

template<typename T>
void DispatchActivationType(ep::Stream* stream, const int64_t m, const int64_t n, 
  const int64_t input_stride, const int64_t output_stride,
  const T* dy, const T* matmul_wx, const T* matmul_vx, 
    T* d_matmul_wx, T* d_matmul_vx
) {
  if (activation == "none") {
    DispatchAlignment<T, ep::primitive::UnaryOp::kIdentity>(stream, m, n, input_stride, output_stride, dy, matmul_wx, matmul_vx, d_matmul_wx, d_matmul_vx);
  } else if (activation == "sigmoid") {
    DispatchAlignment<T, ep::primitive::UnaryOp::kSigmoid>(stream, m, n, input_stride, output_stride, dy, matmul_wx, matmul_vx, d_matmul_wx, d_matmul_vx);
  } else if (activation == "relu") {
    DispatchAlignment<T, ep::primitive::UnaryOp::kRelu>(stream, m, n, input_stride, output_stride, dy, matmul_wx, matmul_vx, d_matmul_wx, d_matmul_vx);
  } else if (activation == "gelu") {
    DispatchAlignment<T, ep::primitive::UnaryOp::kGelu>(stream, m, n, input_stride, output_stride, dy, matmul_wx, matmul_vx, d_matmul_wx, d_matmul_vx);
  } else if (activation == "fast_gelu") {
    DispatchAlignment<T, ep::primitive::UnaryOp::kFastGelu>(stream, m, n, input_stride, output_stride, dy, matmul_wx, matmul_vx, d_matmul_wx, d_matmul_vx);
  } else if (activation == "silu") {
    DispatchAlignment<T, ep::primitive::UnaryOp::kSilu>(stream, m, n, input_stride, output_stride, dy, matmul_wx, matmul_vx, d_matmul_wx, d_matmul_vx);
  } else {
    UNIMPLEMENTED();
  }
}

template<typename T>
class GpuFusedGluWithoutLinearGradKernel final : public user_op::OpKernel {
  public:
    GpuFusedGluWithoutLinearGradKernel() = default;
    ~GpuFusedGluWithoutLinearGradKernel() override = default;

  private:
    using user_op::OpKernel::Compute;
    void Compute(user_op::KernelComputeContext* ctx) const override {
        // obtain tensors from context
        const user_op::Tensor* input_tensor_dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
        const user_op::Tensor* input_tensor_matmul_wx = ctx->Tensor4ArgNameAndIndex("matmul_wx", 0);
        user_op::Tensor* out_tensor_d_matmul_wx = ctx->Tensor4ArgNameAndIndex("d_matmul_wx", 0);

        // obtain optional tensors from context
        bool is_split_mode = false;
        user_op::Tensor* input_tensor_matmul_vx = nullptr;
        user_op::Tensor* out_tensor_d_matmul_vx = nullptr;
        if (ctx->has_input("matmul_vx", 0)){
            input_tensor_matmul_vx = ctx->Tensor4ArgNameAndIndex("matmul_vx", 0);
            out_tensor_d_matmul_vx = ctx->Tensor4ArgNameAndIndex("d_matmul_vx", 0);
            is_split_mode = true;
        }

        // obtain tensor shapes and number of axes
        const ShapeView& dy_shape = input_tensor_dy->shape_view();
        const ShapeView& matmul_wx_shape = input_tensor_matmul_wx->shape_view();
        const ShapeView& d_matmul_wx_shape = out_tensor_d_matmul_wx->shape_view();
        const size_t dy_num_axes = dy_shape.NumAxes();
        const size_t matmul_wx_num_axes = matmul_wx_shape.NumAxes();

        // validate dimension and number of axes
        CHECK_GE_OR_RETURN(dy_num_axes, 2)
          << "number of axes of \'dy\' should have be greater than 1, yet get " << dy_num_axes;
        CHECK_GE_OR_RETURN(matmul_wx_num_axes, 2)
          << "number of axes of \'matmul_wx\' should have be greater than 1, yet get " << matmul_wx_num_axes;
        CHECK_EQ_OR_RETURN(dy_num_axes, matmul_wx_num_axes)
          << "number of axes of \'dy\'(" << dy_num_axes
          << ") is not consistant with the one of \'matmul_wx\'(" << matmul_wx_num_axes
          << ")";
        
        // check input shape
        if(is_split_mode){
            CHECK_EQ_OR_RETURN(2*dy_shape.At(dy_num_axes-1), matmul_wx_shape.At(matmul_wx_num_axes-1))
              << "two times of the last dimension of \'dy\'(" << 2*dy_shape.At(dy_num_axes-1)
              << ") is not consistant with the last dimension of \'matmul_wx\'(" 
              << matmul_wx_shape.At(matmul_wx_num_axes-1) << ")";
        } else {
            CHECK_EQ_OR_RETURN(dy_shape.At(dy_num_axes-1), matmul_wx_shape.At(matmul_wx_num_axes-1))
              << "the last dimension of \'dy\'(" << dy_shape.At(dy_num_axes-1)
              << ") is not consistant with the last dimension of \'matmul_wx\'(" 
              << matmul_wx_shape.At(matmul_wx_num_axes-1) << ")";
        }

        // check optional input tensor shapes
        if(is_split_mode){
            const Shape& matmul_vx_shape = ctx->InputShape("matmul_vx", 0);
            size_t matmul_vx_num_axes = matmul_vx_shape.NumAxes();
            CHECK_GE_OR_RETURN(matmul_vx_num_axes, 2)
              << "number of axes of \'matmul_vx\' should have be greater than 1, yet get " << matmul_vx_num_axes;
            CHECK_EQ_OR_RETURN(matmul_vx_num_axes, dy_num_axes)
              << "number of axes of \'dy\'(" << dy_num_axes
              << ") is not consistant with the one of \'matmul_vx\'(" << matmul_vx_num_axes
              << ")";
            CHECK_EQ_OR_RETURN(matmul_vx_shape.At(matmul_vx_num_axes-1), dy_shape.At(dy_num_axes-1))
              << "the last dimension of \'dy\'(" << dy_shape.At(dy_num_axes-1)
              << ") is not consistant with the last dimension of \'matmul_vx\'(" 
              << matmul_vx_shape.At(matmul_vx_num_axes-1) << ")";
        }

        // infer m, n
        const int64_t m = dy_shape.Count(0, dy_num_axes - 1);
        const int64_t n = dy_shape.At(dy_num_axes - 1);

        // todo: invoke kernel to process
    }

    bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

}

}