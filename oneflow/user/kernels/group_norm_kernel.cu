#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/layer_norm.cuh"
#include <hipcub/hipcub.hpp>

namespace oneflow {

namespace {

template<typename SRC, typename DST, bool affine>
struct AffineStore {
  AffineStore(DST* y, int64_t row_size, int64_t channel_size, int64_t spatial_size,
              const DST* gamma, const DST* beta)
      : y(y),
        row_size(row_size),
        channel_size(channel_size),
        spatial_size(spatial_size),
        gamma(gamma),
        beta(beta) {}

  template<int PackSize>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    cuda::layer_norm::Pack<DST, PackSize> y_pack;
    const int64_t offset = row * row_size + col;
    const int64_t packed_offset = offset / PackSize;
    const int64_t gamma_beta_offset = (offset / spatial_size) % channel_size;
    DST gamma_val = gamma[gamma_beta_offset];
    DST beta_val = beta[gamma_beta_offset];

#pragma unroll
    for (int i = 0; i < PackSize; ++i) {
      DST normalized_i = static_cast<DST>(src[i]);
      if (affine) {
        y_pack.elem[i] = normalized_i * gamma_val + beta_val;
      } else {
        // Direct Store.
        y_pack.elem[i] = normalized_i;
      }
    }
    *(reinterpret_cast<cuda::layer_norm::PackType<DST, PackSize>*>(y) + packed_offset) =
        y_pack.storage;
  }

  DST* y;
  int64_t row_size;
  int64_t channel_size;
  int64_t spatial_size;
  const DST* gamma;
  const DST* beta;
};

template<typename SRC, typename DST, bool affine>
struct ScaleLoad {
  ScaleLoad(const SRC* src, const SRC* gamma, int64_t row_size, int64_t channel_size,
            int64_t spatial_size)
      : src(src),
        gamma(gamma),
        row_size(row_size),
        channel_size(channel_size),
        spatial_size(spatial_size) {}
  template<int PackSize>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    cuda::layer_norm::Pack<SRC, PackSize> src_pack;
    cuda::layer_norm::Pack<SRC, PackSize> gamma_pack;

    const int64_t offset = row * row_size + col;
    const int64_t packed_offset = offset / PackSize;
    const int64_t gamma_offset = (offset / spatial_size) % channel_size;

    src_pack.storage =
        *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, PackSize>*>(src) + packed_offset);
    SRC gamma_val = static_cast<SRC>(1.0);
    if (affine) { gamma_val = gamma[gamma_offset]; }
#pragma unroll
    for (int i = 0; i < PackSize; ++i) { dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_val); }
  }
  const SRC* src;
  const SRC* gamma;
  int64_t row_size;
  int64_t channel_size;
  int64_t spatial_size;
};

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchGroupNormWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon,
                         ComputeType* mean, ComputeType* inv_variance) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                             const int64_t rows, const int64_t cols,
                                             const int64_t spatial_size, const double epsilon,
                                             ComputeType* mean, ComputeType* inv_variance) {
  return DispatchGroupNormWarpImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}

template<typename LOAD, typename STORE, typename ComputeType>
struct TryDispatchGroupNormBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon,
                         ComputeType* mean, ComputeType* inv_variance, bool* success) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType,
                                                                          4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType,
                                                                          2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    } else {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType,
                                                                          1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t TryDispatchGroupNormBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                     const int64_t rows, const int64_t cols,
                                                     const int64_t spatial_size,
                                                     const double epsilon, ComputeType* mean,
                                                     ComputeType* inv_variance, bool* success) {
  return TryDispatchGroupNormBlockSMemImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance, success);
}

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchGroupNormBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon,
                         ComputeType* mean, ComputeType* inv_variance) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                      const int64_t rows, const int64_t cols,
                                                      const int64_t spatial_size,
                                                      const double epsilon, ComputeType* mean,
                                                      ComputeType* inv_variance) {
  return DispatchGroupNormBlockUncachedImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNorm(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                  const int64_t cols, const int64_t spatial_size, const double epsilon,
                  ComputeType* mean, ComputeType* inv_variance) {
  if (cols <= 1024) {
    return DispatchGroupNormWarpImpl<LOAD, STORE, ComputeType>(
        stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchGroupNormBlockSMemImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance,
          &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchGroupNormBlockUncachedImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
    }
    return hipSuccess;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNorm(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                  const int64_t cols, const int64_t spatial_size, const double epsilon,
                  ComputeType* mean, ComputeType* inv_variance) {
  return DispatchGroupNormBlockUncachedImpl<LOAD, STORE, ComputeType>(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}

template<typename T, bool affine>
void GroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                         const int64_t channel_size, const int64_t spatial_size,
                         const double epsilon, const T* x_ptr, const T* gamma_ptr,
                         const T* beta_ptr, T* y_ptr, user_op::Tensor* mean,
                         user_op::Tensor* inv_variance) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, ComputeType> load(x_ptr, norm_size);
  AffineStore<ComputeType, T, affine> store(y_ptr, norm_size, channel_size, spatial_size, gamma_ptr,
                                            beta_ptr);

  DispatchGroupNorm<decltype(load), decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size,
      spatial_size, epsilon, mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchGroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, const int64_t channel_size,
                                 const int64_t spatial_size, const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    GroupNormForwardGpu<T, true>(stream, num_instances, norm_size, channel_size, spatial_size,
                                 epsilon, x_ptr, gamma_ptr, beta_ptr, y_ptr, mean, inv_variance);
  } else {
    GroupNormForwardGpu<T, false>(stream, num_instances, norm_size, channel_size, spatial_size,
                                  epsilon, x_ptr, gamma_ptr, beta_ptr, y_ptr, mean, inv_variance);
  }
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
struct DispatchGroupNormGradWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                         STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                         const int64_t rows, const int64_t cols, const int64_t spatial_size) {
    if (spatial_size % 2 == 0) {
      return cuda::layer_norm::DispatchLayerNormGradWarpImplCols<LOAD_X, LOAD_SCALED_DY, STORE,
                                                                 ComputeType, 2>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    } else {
      return cuda::layer_norm::DispatchLayerNormGradWarpImplCols<LOAD_X, LOAD_SCALED_DY, STORE,
                                                                 ComputeType, 1>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    }
  }
};

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormGradWarpImpl(hipStream_t stream, LOAD_X load_x,
                                                 LOAD_SCALED_DY load_scaled_dy, STORE store,
                                                 const ComputeType* mean,
                                                 const ComputeType* inv_variance,
                                                 const int64_t rows, const int64_t cols,
                                                 const int64_t spatial_size) {
  return DispatchGroupNormGradWarpImplPackSize<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>()(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
struct TryDispatchGroupNormGradBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                         STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                         const int64_t rows, const int64_t cols, const int64_t spatial_size,
                         bool* success) {
    if (spatial_size % 2 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormGradBlockSMemImplBlockSize<
          LOAD_X, LOAD_SCALED_DY, STORE, ComputeType, 2>(stream, load_x, load_scaled_dy, store,
                                                         mean, inv_variance, rows, cols, success);
    } else {
      return cuda::layer_norm::TryDispatchLayerNormGradBlockSMemImplBlockSize<
          LOAD_X, LOAD_SCALED_DY, STORE, ComputeType, 1>(stream, load_x, load_scaled_dy, store,
                                                         mean, inv_variance, rows, cols, success);
    }
  }
};

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline hipError_t TryDispatchGroupNormGradBlockSMemImpl(
    hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy, STORE store,
    const ComputeType* mean, const ComputeType* inv_variance, const int64_t rows,
    const int64_t cols, const int64_t spatial_size, bool* success) {
  return TryDispatchGroupNormGradBlockSMemImplPackSize<LOAD_X, LOAD_SCALED_DY, STORE,
                                                       ComputeType>()(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size, success);
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
struct DispatchGroupNormGradBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                         STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                         const int64_t rows, const int64_t cols, const int64_t spatial_size) {
    if (spatial_size % 2 == 0 && spatial_size > cuda::layer_norm::kWarpSize) {
      return cuda::layer_norm::LaunchLayerNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE,
                                                                    ComputeType, 2>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    } else {
      return cuda::layer_norm::LaunchLayerNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE,
                                                                    ComputeType, 1>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    }
  }
};

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormGradBlockUncachedImpl(hipStream_t stream, LOAD_X load_x,
                                                          LOAD_SCALED_DY load_scaled_dy,
                                                          STORE store, const ComputeType* mean,
                                                          const ComputeType* inv_variance,
                                                          const int64_t rows, const int64_t cols,
                                                          const int64_t spatial_size) {
  return DispatchGroupNormGradBlockUncachedImplPackSize<LOAD_X, LOAD_SCALED_DY, STORE,
                                                        ComputeType>()(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNormGrad(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                      STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                      const int64_t rows, const int64_t cols, const int64_t spatial_size) {
  if (cols <= 1024) {
    return DispatchGroupNormGradWarpImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
        stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err =
          TryDispatchGroupNormGradBlockSMemImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
              stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size,
              &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchGroupNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
    }
    return hipSuccess;
  }
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNormGrad(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                      STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                      const int64_t rows, const int64_t cols, const int64_t spatial_size) {
  return DispatchGroupNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
}

template<typename T, bool affine>
void GroupNormBackwardGpu(ep::Stream* stream, const int64_t num_instances, const int64_t norm_size,
                          const int64_t channel_size, const int64_t spatial_size, const T* dy_ptr,
                          const T* x_ptr, const user_op::Tensor* mean,
                          const user_op::Tensor* inv_variance, const T* gamma_ptr, T* dx_ptr) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, ComputeType> load_x(x_ptr, norm_size);
  ScaleLoad<T, ComputeType, affine> load_scaled_dy(dy_ptr, gamma_ptr, norm_size, channel_size,
                                                   spatial_size);
  cuda::layer_norm::DirectStore<ComputeType, T> store(dx_ptr, norm_size);
  OF_CUDA_CHECK((DispatchGroupNormGrad<decltype(load_x), decltype(load_scaled_dy), decltype(store),
                                       ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
      mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size,
      spatial_size)));
}

template<typename T>
void LaunchGroupNormBackward(ep::Stream* stream, const int64_t num_instances,
                             const int64_t norm_size, const int64_t channel_size,
                             const int64_t spatial_size, const T* dy_ptr, const T* x_ptr,
                             const user_op::Tensor* mean, const user_op::Tensor* inv_variance,
                             const T* gamma_ptr, T* dx_ptr) {
  if (gamma_ptr != nullptr) {
    GroupNormBackwardGpu<T, true>(stream, num_instances, norm_size, channel_size, spatial_size,
                                  dy_ptr, x_ptr, mean, inv_variance, gamma_ptr, dx_ptr);
  } else {
    GroupNormBackwardGpu<T, false>(stream, num_instances, norm_size, channel_size, spatial_size,
                                   dy_ptr, x_ptr, mean, inv_variance, gamma_ptr, dx_ptr);
  }
}

}  // namespace

template<typename T>
class GroupNormGpuKernel final : public user_op::OpKernel {
 public:
  GroupNormGpuKernel() = default;
  ~GroupNormGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0);
    user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    const double epsilon = ctx->Attr<double>("epsilon");
    const int32_t num_groups = ctx->Attr<int32_t>("num_groups");
    CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
    const int64_t num_instances = mean->shape_view().elem_cnt();  // N*num_groups
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const int64_t batch_size = x->shape_view().At(0);
    const int64_t channel_size = x->shape_view().At(1);
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size;
    const T* gamma_ptr = nullptr;
    const T* beta_ptr = nullptr;
    if (ctx->has_input("gamma", 0) && ctx->has_input("beta", 0)) {
      const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
      gamma_ptr = gamma->dptr<T>();
      CHECK_EQ(gamma->shape_view().elem_cnt(), channel_size);
      const user_op::Tensor* beta = ctx->Tensor4ArgNameAndIndex("beta", 0);
      beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>();
      CHECK_EQ(beta->shape_view().elem_cnt(), channel_size);
    }
    DispatchGroupNormForwardGpu<T>(ctx->stream(), num_instances, norm_size, channel_size,
                                   spatial_size, epsilon, x->dptr<T>(), gamma_ptr, beta_ptr,
                                   y->mut_dptr<T>(), mean, inv_variance);
  }
};

#define REGISTER_GROUP_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("group_norm")                                   \
      .SetCreateFn<GroupNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_CUDA_KERNEL(float)
REGISTER_GROUP_NORM_CUDA_KERNEL(double)
#if CUDA_VRSION >= 11000
REGISTER_GROUP_NORM_CUDA_KERNEL(hip_bfloat16)
#endif

template<typename T>
class GroupNormGradGpuKernel final : public user_op::OpKernel {
 public:
  GroupNormGradGpuKernel() = default;
  ~GroupNormGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const int64_t batch_size = x->shape_view().At(0);
    const int64_t channel_size = x->shape_view().At(1);
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size;
    const T* gamma_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      gamma_ptr = ctx->Tensor4ArgNameAndIndex("gamma", 0)->dptr<T>();
    }
    LaunchGroupNormBackward<T>(ctx->stream(), num_instances, norm_size, channel_size, spatial_size,
                               dy->dptr<T>(), x->dptr<T>(), mean, inv_variance, gamma_ptr,
                               dx->mut_dptr<T>());
  };
};

#define REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(dtype)                    \
  REGISTER_USER_KERNEL("group_norm_grad")                              \
      .SetCreateFn<GroupNormGradGpuKernel<dtype>>()                    \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(float)
REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(double)
#if CUDA_VRSION >= 11000
REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(hip_bfloat16)
#endif

constexpr int kBlockSize = 512;
constexpr int kNumWaves = 32;

inline hipError_t GetNumBlocks(int64_t n, int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int tpm;
  {
    hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    if (err != hipSuccess) { return err; }
  }
  *num_blocks = std::max<int>(1, std::min<int64_t>(n, sm_count * tpm / kBlockSize * kNumWaves));
  return hipSuccess;
}

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T, int PackSize>
struct GetPackType {
  using type = typename std::aligned_storage<sizeof(T) * PackSize, sizeof(T) * PackSize>::type;
};

template<typename T, int PackSize>
using PackType = typename GetPackType<T, PackSize>::type;

template<typename T, int PackSize>
union Pack {
  static_assert(sizeof(PackType<T, PackSize>) == sizeof(T) * PackSize, "");
  __device__ Pack() {
    // do nothing
  }

  __device__ Pack<T, PackSize> operator*(Pack<T, PackSize> pack) {
    Pack<T, PackSize> newPack;
#pragma unroll
    for (int i = 0; i < PackSize; i++) { newPack.elem[i] = elem[i] * pack.elem[i]; }
    return newPack;
  }

  T elem[PackSize];
  PackType<T, PackSize> storage;
};

template<typename ComputeType, typename T, int PackSize>
__device__ ComputeType PackReduce(Pack<T, PackSize> pack) {
  ComputeType result = 0.0;
#pragma unroll
  for (int i = 0; i < PackSize; i++) { result += static_cast<ComputeType>(pack.elem[i]); }
  return result;
}

constexpr int kMaxPackBytes = 128 / 8;
constexpr int kMaxPackSize = 8;

constexpr int Min(int a, int b) { return a < b ? a : b; }

template<typename T>
constexpr int GetPackSize() {
  return Min(kMaxPackBytes / sizeof(T), kMaxPackSize);
}

template<typename T, typename ComputeType, int PackSize>
__global__ void GroupNormParamGradKernel(const T* dy, const T* x, const T* mean, const T* inv_var,
                                         T* dgamma, T* dbeta, const int32_t batch_size,
                                         const int32_t group_size, const int32_t channel_size,
                                         const int32_t spatial_size) {
  using LoadType = PackType<T, PackSize>;
  for (int32_t channel = blockIdx.x; channel < channel_size; channel += gridDim.x) {
    ComputeType dgamma_sum = 0.0;
    ComputeType dbeta_sum = 0.0;
    const int32_t group_num = channel_size / group_size;
    for (int32_t batch = 0; batch < batch_size; batch++) {
      const int32_t batch_channel_id = batch * channel_size + channel;
      ComputeType ds_sum = 0.0;
      ComputeType db_sum = 0.0;
      for (int32_t spatial = threadIdx.x * PackSize; spatial < spatial_size;
           spatial += blockDim.x * PackSize) {
        Pack<T, PackSize> dy_pack{};
        Pack<T, PackSize> x_pack{};
        const int32_t load_idx = batch_channel_id * spatial_size + spatial;
        const LoadType* dy_load = reinterpret_cast<const LoadType*>(dy + load_idx);
        dy_pack.storage = *dy_load;
        const LoadType* x_load = reinterpret_cast<const LoadType*>(x + load_idx);
        x_pack.storage = *x_load;
        ds_sum += PackReduce<ComputeType, T, PackSize>(dy_pack * x_pack);
        db_sum += PackReduce<ComputeType, T, PackSize>(dy_pack);
      }
      const int32_t batch_group_id = batch * group_size + channel / group_num;
      ComputeType mean_val = static_cast<ComputeType>(mean[batch_group_id]);
      ComputeType inv_var_val = static_cast<ComputeType>(inv_var[batch_group_id]);
      dgamma_sum += (ds_sum - db_sum * mean_val) * inv_var_val;
      dbeta_sum += db_sum;
    }
    __syncthreads();
    typedef hipcub::BlockReduce<ComputeType, kBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage1;
    __shared__ typename BlockReduce::TempStorage temp_storage2;
    ComputeType dgamma_sum_result =
        BlockReduce(temp_storage1).Reduce(dgamma_sum, SumOp<ComputeType>());
    ComputeType dbeta_sum_result =
        BlockReduce(temp_storage2).Reduce(dbeta_sum, SumOp<ComputeType>());
    if (threadIdx.x == 0) {
      dgamma[channel] = dgamma_sum_result;
      dbeta[channel] = dbeta_sum_result;
    }
  }
}

template<typename T>
int32_t GetLaunchPackSize(const int32_t spatial_size) {
  for (int pack_size = GetPackSize<T>(); pack_size > 0; pack_size /= 2) {
    if (spatial_size % pack_size == 0) { return pack_size; }
  }
  return 1;
}

template<typename T, typename ComputeType>
void DispatchGroupNormParamGradKernel(ep::Stream* stream, const T* dy, const T* x, const T* mean,
                                      const T* inv_var, T* dgamma, T* dbeta,
                                      const int32_t batch_size, const int32_t group_size,
                                      const int32_t channel_size, const int32_t spatial_size) {
  const int launch_pack_size = GetLaunchPackSize<T>(spatial_size);
  int num_blocks;
  OF_CUDA_CHECK(GetNumBlocks(channel_size, &num_blocks));
  if (launch_pack_size == 8) {
    GroupNormParamGradKernel<T, ComputeType, 8>
        <<<num_blocks, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, dgamma, dbeta, batch_size, group_size, channel_size,
            spatial_size);
  } else if (launch_pack_size == 4) {
    GroupNormParamGradKernel<T, ComputeType, 4>
        <<<num_blocks, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, dgamma, dbeta, batch_size, group_size, channel_size,
            spatial_size);
  } else if (launch_pack_size == 2) {
    GroupNormParamGradKernel<T, ComputeType, 2>
        <<<num_blocks, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, dgamma, dbeta, batch_size, group_size, channel_size,
            spatial_size);
  } else {
    GroupNormParamGradKernel<T, ComputeType, 1>
        <<<num_blocks, kBlockSize, 0, stream->As<ep::CudaStream>()->hip_stream()>>>(
            dy, x, mean, inv_var, dgamma, dbeta, batch_size, group_size, channel_size,
            spatial_size);
  }
}

template<typename T>
class GroupNormParamGradGpuKernel final : public user_op::OpKernel {
 public:
  GroupNormParamGradGpuKernel() = default;
  ~GroupNormParamGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dgamma = ctx->Tensor4ArgNameAndIndex("dgamma", 0);
    user_op::Tensor* dbeta = ctx->Tensor4ArgNameAndIndex("dbeta", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    const int64_t batch_size = x->shape_view().At(0);
    const int64_t channel_size = x->shape_view().At(1);
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size;
    const int64_t group_size = num_instances / batch_size;
    using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
    DispatchGroupNormParamGradKernel<T, ComputeType>(
        ctx->stream(), dy->dptr<T>(), x->dptr<T>(), mean->dptr<T>(), inv_variance->dptr<T>(),
        dgamma->mut_dptr<T>(), dbeta->mut_dptr<T>(), batch_size, group_size, channel_size,
        spatial_size);
  };
};

#define REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(dtype)              \
  REGISTER_USER_KERNEL("group_norm_param_grad")                        \
      .SetCreateFn<GroupNormParamGradGpuKernel<dtype>>()               \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

// REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(float)
// REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(double)
// #if CUDA_VRSION >= 11000
// REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(hip_bfloat16)
// #endif

}  // namespace oneflow
