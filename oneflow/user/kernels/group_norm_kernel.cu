#include "hip/hip_runtime.h"
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ndarray/ndarray_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/cuda/layer_norm.cuh"
#include <hipcub/hipcub.hpp>

namespace oneflow{

namespace {

// TODO add AFFINE STORE

template<typename SRC, typename DST, bool affine>
struct AffineStore{
    AffineStore(DST* y, int64_t row_size, int64_t channel_size, int64_t spatial_size, const DST* gamma, const DST* beta)
    : y(y), row_size(row_size), channel_size(channel_size), spatial_size(spatial_size), gamma(gamma), beta(beta) {}

    template<int PackSize>
    __device__ void store(const SRC* src, int64_t row, int64_t col){
        cuda::layer_norm::Pack<DST, PackSize> y_pack;
        const int64_t offset = row * row_size + col; 
        const int64_t packed_offset = offset / PackSize;
        const int64_t gamma_beta_offset = (offset / spatial_size) % channel_size;
        DST gamma_val = gamma[gamma_beta_offset]; 
        DST beta_val = beta[gamma_beta_offset]; 

    #pragma unroll
        for (int i = 0; i < PackSize; ++i) {
            DST normalized_i = static_cast<DST>(src[i]);
            if(affine){
                y_pack.elem[i] = normalized_i * gamma_val + beta_val;
            } else {
                // Direct Store. 
                y_pack.elem[i] = normalized_i; 
            }
        }
        *(reinterpret_cast<cuda::layer_norm::PackType<DST, PackSize>*>(y) + packed_offset) = y_pack.storage;
    }

    DST* y;
    int64_t row_size;
    int64_t channel_size;
    int64_t spatial_size;
    const DST* gamma;
    const DST* beta;
}; 

template<typename SRC, typename DST, bool affine>
struct ScaleLoad {
  ScaleLoad(const SRC* src, const SRC* gamma, int64_t row_size, int64_t channel_size, int64_t spatial_size)
      : src(src), gamma(gamma), row_size(row_size), channel_size(channel_size), spatial_size(spatial_size) {}
  template<int PackSize>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    cuda::layer_norm::Pack<SRC, PackSize> src_pack;
    cuda::layer_norm::Pack<SRC, PackSize> gamma_pack;

    const int64_t offset = row * row_size + col; 
    const int64_t packed_offset = offset / PackSize;
    const int64_t gamma_offset = (offset / spatial_size) % channel_size;

    src_pack.storage = *(reinterpret_cast<const cuda::layer_norm::PackType<SRC, PackSize>*>(src) + packed_offset);
    SRC gamma_val = static_cast<SRC>(1.0); 
    // if (affine) {
    //   gamma_val = gamma[gamma_offset]; 
    // } 
#pragma unroll
    for (int i = 0; i < PackSize; ++i) {
      dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_val);
    }
  }
  const SRC* src;
  const SRC* gamma;
  int64_t row_size;
  int64_t channel_size;
  int64_t spatial_size;
};

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchGroupNormWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                         ComputeType* inv_variance) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else {
      return cuda::layer_norm::DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormWarpImpl(hipStream_t stream, 
                                             LOAD load, STORE store,
                                             const int64_t rows, const int64_t cols,
                                             const int64_t spatial_size, 
                                             const double epsilon, ComputeType* mean,
                                             ComputeType* inv_variance) {
  return DispatchGroupNormWarpImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}


template<typename LOAD, typename STORE, typename ComputeType>
struct TryDispatchGroupNormBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                         ComputeType* inv_variance, bool* success) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    } else {
      return cuda::layer_norm::TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance, success);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t TryDispatchGroupNormBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                     const int64_t rows, const int64_t cols, 
                                                     const int64_t spatial_size, 
                                                     const double epsilon, ComputeType* mean,
                                                     ComputeType* inv_variance, bool* success) {
  return TryDispatchGroupNormBlockSMemImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance, success);
}

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchGroupNormBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                         ComputeType* inv_variance) {
    if (spatial_size % 4 == 0) {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else if (spatial_size % 2 == 0) {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    } else {
      return cuda::layer_norm::LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, mean, inv_variance);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                      const int64_t rows, const int64_t cols,
                                                      const int64_t spatial_size, 
                                                      const double epsilon, ComputeType* mean,
                                                      ComputeType* inv_variance) {
  return DispatchGroupNormBlockUncachedImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
}


template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNorm(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                  const int64_t cols, const int64_t spatial_size, const double epsilon, ComputeType* mean,
                  ComputeType* inv_variance) {
  if (cols <= 1024) {
    return DispatchGroupNormWarpImpl<LOAD, STORE, ComputeType>(stream, load, store, rows, cols, spatial_size, 
                                                               epsilon, mean, inv_variance);
  } else {
    // TODO
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchGroupNormBlockSMemImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance,
          &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchGroupNormBlockUncachedImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, spatial_size, epsilon, mean, inv_variance);
    }
    return hipSuccess;
  }
}

template<typename T, bool affine>
void GroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances, 
                        const int64_t norm_size, 
                        const int64_t channel_size, 
                        const int64_t spatial_size, 
                        const double epsilon, const T* x_ptr, const T* gamma_ptr,
                        const T* beta_ptr, T* y_ptr, user_op::Tensor* mean,
                        user_op::Tensor* inv_variance) {
    using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
    cuda::layer_norm::DirectLoad<T, ComputeType> load(x_ptr, norm_size);
    AffineStore<ComputeType, T, affine> store(y_ptr, norm_size, channel_size, spatial_size, gamma_ptr, beta_ptr);

    DispatchGroupNorm<decltype(load), decltype(store), ComputeType>(
        stream->As<ep::CudaStream>()->hip_stream(), load, store, num_instances, norm_size, spatial_size, 
        epsilon, mean->mut_dptr<ComputeType>(), inv_variance->mut_dptr<ComputeType>());
}

template<typename T>
void DispatchGroupNormForwardGpu(ep::Stream* stream, const int64_t num_instances,
                                 const int64_t norm_size, 
                                 const int64_t channel_size, 
                                 const int64_t spatial_size, 
                                 const double epsilon, const T* x_ptr,
                                 const T* gamma_ptr, const T* beta_ptr, T* y_ptr,
                                 user_op::Tensor* mean, user_op::Tensor* inv_variance) {
  if (gamma_ptr != nullptr && beta_ptr != nullptr) {
    GroupNormForwardGpu<T, true>(stream, num_instances, norm_size, channel_size, spatial_size, epsilon, x_ptr, gamma_ptr,
                                       beta_ptr, y_ptr, mean, inv_variance);
  } else {
    GroupNormForwardGpu<T, false>(stream, num_instances, norm_size, channel_size, spatial_size, epsilon, x_ptr,
                                         gamma_ptr, beta_ptr, y_ptr, mean, inv_variance);
  }
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
struct DispatchGroupNormGradWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                         STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                         const int64_t rows, const int64_t cols, const int64_t spatial_size) {
    if (spatial_size % 2 == 0) {
      return cuda::layer_norm::DispatchLayerNormGradWarpImplCols<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType, 2>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    } else {
      return cuda::layer_norm::DispatchLayerNormGradWarpImplCols<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType, 1>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    }
  }
};

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormGradWarpImpl(hipStream_t stream, LOAD_X load_x,
                                                 LOAD_SCALED_DY load_scaled_dy, STORE store,
                                                 const ComputeType* mean,
                                                 const ComputeType* inv_variance,
                                                 const int64_t rows, const int64_t cols, 
                                                 const int64_t spatial_size) {
  return DispatchGroupNormGradWarpImplPackSize<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>()(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
struct TryDispatchGroupNormGradBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                         STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                         const int64_t rows, const int64_t cols, const int64_t spatial_size, 
                         bool* success) {
    if (spatial_size % 2 == 0) {
      return cuda::layer_norm::TryDispatchLayerNormGradBlockSMemImplBlockSize<LOAD_X, LOAD_SCALED_DY, STORE,
                                                            ComputeType, 2>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, success);
    } else {
      return cuda::layer_norm::TryDispatchLayerNormGradBlockSMemImplBlockSize<LOAD_X, LOAD_SCALED_DY, STORE,
                                                            ComputeType, 1>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, success);
    }
  }
};


template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline hipError_t TryDispatchGroupNormGradBlockSMemImpl(hipStream_t stream, LOAD_X load_x,
                                                         LOAD_SCALED_DY load_scaled_dy, STORE store,
                                                         const ComputeType* mean,
                                                         const ComputeType* inv_variance,
                                                         const int64_t rows, const int64_t cols,
                                                         const int64_t spatial_size, 
                                                         bool* success) {
  return TryDispatchGroupNormGradBlockSMemImplPackSize<LOAD_X, LOAD_SCALED_DY, STORE,
                                                       ComputeType>()(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size, success);
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
struct DispatchGroupNormGradBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                         STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                         const int64_t rows, const int64_t cols, const int64_t spatial_size) {
    if (spatial_size % 2 == 0 && spatial_size > cuda::layer_norm::kWarpSize) {
      return cuda::layer_norm::LaunchLayerNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType, 2>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    } else {
      return cuda::layer_norm::LaunchLayerNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType, 1>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols);
    }
  }
};

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline hipError_t DispatchGroupNormGradBlockUncachedImpl(hipStream_t stream, LOAD_X load_x,
                                                          LOAD_SCALED_DY load_scaled_dy,
                                                          STORE store, const ComputeType* mean,
                                                          const ComputeType* inv_variance,
                                                          const int64_t rows, const int64_t cols, 
                                                          const int64_t spatial_size) {
  return DispatchGroupNormGradBlockUncachedImplPackSize<LOAD_X, LOAD_SCALED_DY, STORE,
                                                        ComputeType>()(
      stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
}

template<typename LOAD_X, typename LOAD_SCALED_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchGroupNormGrad(hipStream_t stream, LOAD_X load_x, LOAD_SCALED_DY load_scaled_dy,
                      STORE store, const ComputeType* mean, const ComputeType* inv_variance,
                      const int64_t rows, const int64_t cols, const int64_t spatial_size) {
  if (cols <= 1024) {
    return DispatchGroupNormGradWarpImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
        stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err =
          TryDispatchGroupNormGradBlockSMemImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
              stream, load_x, load_scaled_dy, store, mean, inv_variance, 
              rows, cols, spatial_size, 
              &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchGroupNormGradBlockUncachedImpl<LOAD_X, LOAD_SCALED_DY, STORE, ComputeType>(
          stream, load_x, load_scaled_dy, store, mean, inv_variance, rows, cols, spatial_size);
    }
    return hipSuccess;
  }
}


template<typename T, bool affine>
void GroupNormBackwardGpu(ep::Stream* stream, const int64_t num_instances, 
                          const int64_t norm_size, const int64_t channel_size, const int64_t spatial_size, 
                          const T* dy_ptr, const T* x_ptr, const user_op::Tensor* mean,
                          const user_op::Tensor* inv_variance, const T* gamma_ptr,
                          T* dx_ptr) {
  using ComputeType = typename cuda::layer_norm::DefaultComputeType<T>::type;
  cuda::layer_norm::DirectLoad<T, ComputeType> load_x(x_ptr, norm_size);
  ScaleLoad<T, ComputeType, affine> load_scaled_dy(dy_ptr, gamma_ptr, norm_size, channel_size, spatial_size);
  cuda::layer_norm::DirectStore<ComputeType, T> store(dx_ptr, norm_size);
  OF_CUDA_CHECK((DispatchGroupNormGrad<decltype(load_x), decltype(load_scaled_dy),
                                       decltype(store), ComputeType>(
      stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
      mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size, spatial_size)));

  // OF_CUDA_CHECK((cuda::layer_norm::DispatchLayerNormGrad<decltype(load_x), decltype(load_scaled_dy),
  //                                                        decltype(store), ComputeType>(
  //     stream->As<ep::CudaStream>()->hip_stream(), load_x, load_scaled_dy, store,
  //     mean->dptr<ComputeType>(), inv_variance->dptr<ComputeType>(), num_instances, norm_size)));

}

template<typename T>
void LaunchGroupNormBackward(ep::Stream* stream, const int64_t num_instances,
                             const int64_t norm_size, const int64_t channel_size, 
                             const int64_t spatial_size, 
                             const T* dy_ptr, const T* x_ptr,
                             const user_op::Tensor* mean, const user_op::Tensor* inv_variance,
                             const T* gamma_ptr, T* dx_ptr) {
  if (gamma_ptr != nullptr) {
    GroupNormBackwardGpu<T, true>(stream, num_instances, norm_size, channel_size, 
                                  spatial_size, dy_ptr, x_ptr, mean,
                                  inv_variance, gamma_ptr, dx_ptr);
  } else {
    GroupNormBackwardGpu<T, false>(stream, num_instances, norm_size, channel_size, spatial_size, 
                                    dy_ptr, x_ptr, mean,
                                    inv_variance, gamma_ptr, dx_ptr);
  }
}


} // namespace 

template<typename T>
class GroupNormGpuKernel final : public user_op::OpKernel{

public: 
    GroupNormGpuKernel() = default; 
    ~GroupNormGpuKernel() = default; 

private: 
    using user_op::OpKernel::Compute; 
    bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
    void Compute(user_op::KernelComputeContext* ctx) const override {
        const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0); 
        user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("y", 0); 
        user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
        user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
        const double epsilon = ctx->Attr<double>("epsilon");
        const int32_t num_groups = ctx->Attr<int32_t>("num_groups"); 
        CHECK_GE(epsilon, HIPDNN_BN_MIN_EPSILON);
        const int64_t num_instances = mean->shape_view().elem_cnt();  // N*num_groups
        const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
        const int64_t batch_size = x->shape_view().At(0); 
        const int64_t channel_size = x->shape_view().At(1); 
        const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size; 
        const T* gamma_ptr = nullptr;
        const T* beta_ptr = nullptr;
        if (ctx->has_input("gamma", 0) && ctx->has_input("beta", 0)) {
          const user_op::Tensor* gamma = ctx->Tensor4ArgNameAndIndex("gamma", 0);
          gamma_ptr = gamma->dptr<T>();
          CHECK_EQ(gamma->shape_view().elem_cnt(), channel_size);
          const user_op::Tensor* beta = ctx->Tensor4ArgNameAndIndex("beta", 0); 
          beta_ptr = ctx->Tensor4ArgNameAndIndex("beta", 0)->dptr<T>();
          CHECK_EQ(beta->shape_view().elem_cnt(), channel_size);
        }
        DispatchGroupNormForwardGpu<T>(ctx->stream(), 
                                       num_instances, norm_size, 
                                       channel_size, 
                                       spatial_size, epsilon, 
                                       x->dptr<T>(),
                                       gamma_ptr, beta_ptr, y->mut_dptr<T>(), mean, inv_variance);

    }

}; 

#define REGISTER_GROUP_NORM_CUDA_KERNEL(dtype)                         \
  REGISTER_USER_KERNEL("group_norm")                                   \
      .SetCreateFn<GroupNormGpuKernel<dtype>>()                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("x", 0) == GetDataType<dtype>::value));

// REGISTER_GROUP_NORM_CUDA_KERNEL(half)
REGISTER_GROUP_NORM_CUDA_KERNEL(float)
// REGISTER_GROUP_NORM_CUDA_KERNEL(double)

template<typename T>
class GroupNormGradGpuKernel final : public user_op::OpKernel {
 public:
  GroupNormGradGpuKernel() = default;
  ~GroupNormGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    printf("Num instances is: %ld \n", num_instances); 
    printf("Norm size is: %ld \n", norm_size); 

    const int64_t batch_size = x->shape_view().At(0); 
    const int64_t channel_size = x->shape_view().At(1); 
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size; 
    const T* gamma_ptr = nullptr;
    if (ctx->has_input("gamma", 0)) {
      gamma_ptr = ctx->Tensor4ArgNameAndIndex("gamma", 0)->dptr<T>();
    }
    LaunchGroupNormBackward<T>(ctx->stream(), num_instances, norm_size, channel_size, spatial_size, 
                               dy->dptr<T>(), x->dptr<T>(),
                               mean, inv_variance, gamma_ptr, dx->mut_dptr<T>());
                               
  };
};

#define REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(dtype)                                        \
  REGISTER_USER_KERNEL("group_norm_grad")                                                  \
      .SetCreateFn<GroupNormGradGpuKernel<dtype>>()                                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_GRAD_CUDA_KERNEL(float)


// template<typename T, typename ComputeType>
// __global__ void GroupNormParamGradKernel(const T* dy, 
//                                          const T* x, 
//                                          const T* mean, 
//                                          const T* inv_var, 
//                                          T* dgamma, 
//                                          T* dbeta, 
//                                          const int32_t batch_size, 
//                                          const int32_t group_size, 
//                                          const int32_t channel_size, 
//                                          const int32_t spatial_size){
//   // Assume each thread compute each norm
//   /*
//   actually: n, g, c // g, h, w
//   mean: n, g
//   alpha: c
//   */
//   const int32_t global_thread_idx = blockDim.x * blockIdx.x + threadIdx.x; 
//   const int32_t step = gridDim.x * blockDim.x; 
//   for(int32_t channel=global_thread_idx; channel < channel_size; channel+=step){
//     ComputeType dgamma_sum = 0.0; 
//     ComputeType dbeta_sum = 0.0; 
//     const int32_t D = channel_size / group_size; 
//     for(int32_t batch=0; batch < batch_size; batch++){
//       const int32_t batch_channel_id = batch * channel_size + channel; 
//       ComputeType ds_sum = 0.0; 
//       ComputeType db_sum = 0.0; 
//       for(int32_t spatial=0; spatial < spatial_size; spatial++){
//         ComputeType dy_val = static_cast<ComputeType>(dy[batch_channel_id * spatial_size + spatial]); 
//         ComputeType x_val = static_cast<ComputeType>(x[batch_channel_id * spatial_size + spatial]); 
//         ds_sum += dy_val * x_val; 
//         db_sum += dy_val; 
//       }
//       const int32_t batch_group_id = batch * group_size + channel / D; 
//       ComputeType mean_val = static_cast<ComputeType>(mean[batch_group_id]); 
//       ComputeType inv_var_val = static_cast<ComputeType>(inv_var[batch_group_id]); 
//       dgamma_sum += (ds_sum - db_sum * mean_val) * inv_var_val; 
//       dbeta_sum += db_sum;
//     }
//     dgamma[channel] = dgamma_sum; 
//     dbeta[channel] = dbeta_sum; 
//   }
// }

constexpr int kBlockSize = 128; 

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T, typename ComputeType>
__global__ void GroupNormParamGradKernel(const T* dy, 
                                         const T* x, 
                                         const T* mean, 
                                         const T* inv_var, 
                                         T* dgamma, 
                                         T* dbeta, 
                                         const int32_t batch_size, 
                                         const int32_t group_size, 
                                         const int32_t channel_size, 
                                         const int32_t spatial_size){
  for(int32_t channel=blockIdx.x; channel < channel_size; channel+=gridDim.x){
    ComputeType dgamma_sum = 0.0; 
    ComputeType dbeta_sum = 0.0; 
    const int32_t D = channel_size / group_size; 
    for(int32_t batch=0; batch < batch_size; batch++){
      const int32_t batch_channel_id = batch * channel_size + channel; 
      ComputeType ds_sum = 0.0; 
      ComputeType db_sum = 0.0; 
      for(int32_t spatial=threadIdx.x; spatial < spatial_size; spatial+=blockDim.x){
        ComputeType dy_val = static_cast<ComputeType>(dy[batch_channel_id * spatial_size + spatial]); 
        ComputeType x_val = static_cast<ComputeType>(x[batch_channel_id * spatial_size + spatial]); 
        ds_sum += dy_val * x_val; 
        db_sum += dy_val; 
      }
      const int32_t batch_group_id = batch * group_size + channel / D; 
      ComputeType mean_val = static_cast<ComputeType>(mean[batch_group_id]); 
      ComputeType inv_var_val = static_cast<ComputeType>(inv_var[batch_group_id]); 
      dgamma_sum += (ds_sum - db_sum * mean_val) * inv_var_val; 
      dbeta_sum += db_sum;
    }
    __syncthreads(); 
    typedef hipcub::BlockReduce<ComputeType, kBlockSize> BlockReduce; 
    __shared__ typename BlockReduce::TempStorage temp_storage1;
    __shared__ typename BlockReduce::TempStorage temp_storage2;
    ComputeType dgamma_sum_result = BlockReduce(temp_storage1).Reduce(dgamma_sum, SumOp<ComputeType>());
    ComputeType dbeta_sum_result = BlockReduce(temp_storage2).Reduce(dbeta_sum, SumOp<ComputeType>());
    if(threadIdx.x == 0){
      dgamma[channel] = dgamma_sum_result; 
      dbeta[channel] = dbeta_sum_result;
    }
  }
}

template<typename T>
class GroupNormParamGradGpuKernel final : public user_op::OpKernel {
 public:
  GroupNormParamGradGpuKernel() = default;
  ~GroupNormParamGradGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("x", 0);
    const user_op::Tensor* mean = ctx->Tensor4ArgNameAndIndex("mean", 0);
    const user_op::Tensor* inv_variance = ctx->Tensor4ArgNameAndIndex("inv_variance", 0);
    user_op::Tensor* dgamma = ctx->Tensor4ArgNameAndIndex("dgamma", 0);
    user_op::Tensor* dbeta = ctx->Tensor4ArgNameAndIndex("dbeta", 0);

    /*
    actually: n, g, c // g, h, w
    mean: n, g
    alpha: c

    */

    const int64_t num_instances = mean->shape_view().elem_cnt();
    const int64_t norm_size = x->shape_view().elem_cnt() / num_instances;
    printf("Num instances is: %ld \n", num_instances); 
    printf("Norm size is: %ld \n", norm_size); 

    const int64_t batch_size = x->shape_view().At(0); 
    const int64_t channel_size = x->shape_view().At(1); 
    const int64_t spatial_size = x->shape_view().elem_cnt() / batch_size / channel_size; 
    const int64_t group_size = num_instances / batch_size; 
    printf("batch_size is: %ld \n", batch_size); 
    printf("channel_size is: %ld \n", channel_size); 
    printf("spatial_size is: %ld \n", spatial_size); 
    printf("group_size is: %ld \n", group_size); 
    const int32_t grid_size = channel_size; 
    GroupNormParamGradKernel<T, float><<<grid_size, kBlockSize, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(dy->dptr<T>(), 
                                                                                                       x->dptr<T>(), 
                                                                                                       mean->dptr<T>(), 
                                                                                                       inv_variance->dptr<T>(), 
                                                                                                       dgamma->mut_dptr<T>(), 
                                                                                                       dbeta->mut_dptr<T>(), 
                                                                                                       batch_size, 
                                                                                                       group_size, 
                                                                                                       channel_size, 
                                                                                                       spatial_size); 
  };
};

#define REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(dtype)                                        \
  REGISTER_USER_KERNEL("group_norm_param_grad")                                                  \
      .SetCreateFn<GroupNormParamGradGpuKernel<dtype>>()                                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                     \
                       && (user_op::HobDataType("dy", 0) == GetDataType<dtype>::value));

REGISTER_GROUP_NORM_PARAM_GRAD_CUDA_KERNEL(float)



} // namespace oneflow 