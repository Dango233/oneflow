#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/ep/include/device.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/user/kernels/distributions/common.h"
#include "oneflow/user/kernels/random_seed_util.h"
#include "oneflow/user/kernels/cuda_macros.h"

// NOTE(Liang Depeng): the implementation of BernoulliScalarGpuKerenl is modified from
// https://github.com/pytorch/pytorch/blob/master/aten/src/ATen/native/cuda/DistributionTemplates.h
namespace oneflow {

namespace {

OF_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void distribution_elementwise_grid_stride_kernel_double(int32_t numel, uint64_t seed,
                                                                   uint64_t offset,
                                                                   const double* p_dptr,
                                                                   double* out_ptr) {
  int32_t unroll_factor = 2;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int rounded_size = ((numel - 1) / (blockDim.x * gridDim.x * unroll_factor) + 1) * blockDim.x
                     * gridDim.x * unroll_factor;
  for (int32_t linear_index = idx; linear_index < rounded_size;
       linear_index += blockDim.x * gridDim.x * unroll_factor) {
    double2 rand = hiprand_uniform2_double(&state);
#pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) { out_ptr[li] = static_cast<double>((&rand.x)[ii] < p_dptr[li]); }
    }
    __syncthreads();
  }
}

OF_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void distribution_elementwise_grid_stride_kernel_float(int32_t numel, uint64_t seed,
                                                                  uint64_t offset,
                                                                  const float* p_dptr,
                                                                  float* out_ptr) {
  int32_t unroll_factor = 4;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int rounded_size = ((numel - 1) / (blockDim.x * gridDim.x * unroll_factor) + 1) * blockDim.x
                     * gridDim.x * unroll_factor;
  for (int32_t linear_index = idx; linear_index < rounded_size;
       linear_index += blockDim.x * gridDim.x * unroll_factor) {
    float4 rand = hiprand_uniform4(&state);
#pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) { out_ptr[li] = static_cast<float>((&rand.x)[ii] < p_dptr[li]); }
    }
    __syncthreads();
  }
}

}  // namespace

class BernoulliTensorGpuFloatKerenl final : public user_op::OpKernel {
 public:
  BernoulliTensorGpuFloatKerenl() = default;
  ~BernoulliTensorGpuFloatKerenl() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const auto& generator = CHECK_JUST(one::MakeGenerator(DeviceType::kCUDA));
    generator->set_current_seed(ctx->Attr<int64_t>("seed"));
    return std::make_shared<DistributionKernelState>(generator);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    const user_op::Tensor* p_blob = ctx->Tensor4ArgNameAndIndex("p", 0);
    user_op::Tensor* out_blob = ctx->Tensor4ArgNameAndIndex("out", 0);
    const float* p_dptr = p_blob->dptr<float>();
    float* out_dptr = out_blob->mut_dptr<float>();
    const int64_t elem_cnt = out_blob->shape_view().elem_cnt();
    CHECK_GT(elem_cnt, 0);

    auto* kernel_state = dynamic_cast<DistributionKernelState*>(state);
    CHECK_NOTNULL(kernel_state);
    const auto& generator = kernel_state->generator();
    CHECK_NOTNULL(generator);
    const auto& gpu_generator = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>());

    ep::CudaStream* hip_stream = ctx->stream()->As<ep::CudaStream>();
    auto execution_policy = calc_execution_policy(elem_cnt, hip_stream);

    auto counter_offset = std::get<0>(execution_policy);
    auto grid = std::get<1>(execution_policy);
    auto block = std::get<2>(execution_policy);

    uint64_t offset = 0;
    uint64_t seed = gpu_generator->current_seed();
    {
      std::lock_guard<std::mutex> lock(gpu_generator->mutex_);
      offset = gpu_generator->get_philox_offset(counter_offset);
    }

    distribution_elementwise_grid_stride_kernel_float<<<
        grid, block, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        elem_cnt, seed, offset, p_dptr, out_dptr);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

class BernoulliTensorGpuDoubleKerenl final : public user_op::OpKernel {
 public:
  BernoulliTensorGpuDoubleKerenl() = default;
  ~BernoulliTensorGpuDoubleKerenl() override = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const auto& generator = CHECK_JUST(one::MakeGenerator(DeviceType::kCUDA));
    generator->set_current_seed(ctx->Attr<int64_t>("seed"));
    return std::make_shared<DistributionKernelState>(generator);
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state,
               const user_op::OpKernelCache*) const override {
    const user_op::Tensor* p_blob = ctx->Tensor4ArgNameAndIndex("p", 0);
    user_op::Tensor* out_blob = ctx->Tensor4ArgNameAndIndex("out", 0);
    const double* p_dptr = p_blob->dptr<double>();
    double* out_dptr = out_blob->mut_dptr<double>();
    const int64_t elem_cnt = out_blob->shape_view().elem_cnt();
    CHECK_GT(elem_cnt, 0);

    auto* kernel_state = dynamic_cast<DistributionKernelState*>(state);
    CHECK_NOTNULL(kernel_state);
    const auto& generator = kernel_state->generator();
    CHECK_NOTNULL(generator);
    const auto& gpu_generator = CHECK_JUST(generator->Get<one::CUDAGeneratorImpl>());

    ep::CudaStream* hip_stream = ctx->stream()->As<ep::CudaStream>();
    auto execution_policy = calc_execution_policy(elem_cnt, hip_stream);

    auto counter_offset = std::get<0>(execution_policy);
    auto grid = std::get<1>(execution_policy);
    auto block = std::get<2>(execution_policy);

    uint64_t offset = 0;
    uint64_t seed = gpu_generator->current_seed();
    {
      std::lock_guard<std::mutex> lock(gpu_generator->mutex_);
      offset = gpu_generator->get_philox_offset(counter_offset);
    }

    distribution_elementwise_grid_stride_kernel_double<<<
        grid, block, 0, ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        elem_cnt, seed, offset, p_dptr, out_dptr);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

REGISTER_USER_KERNEL("bernoulli_tensor")
    .SetCreateFn<BernoulliTensorGpuFloatKerenl>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobDataType("p", 0) == GetDataType<float>::value)
                     && (user_op::HobDataType("out", 0) == GetDataType<float>::value));

REGISTER_USER_KERNEL("bernoulli_tensor")
    .SetCreateFn<BernoulliTensorGpuDoubleKerenl>()
    .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)
                     && (user_op::HobDataType("p", 0) == GetDataType<double>::value)
                     && (user_op::HobDataType("out", 0) == GetDataType<double>::value));

}  // namespace oneflow
